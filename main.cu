#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <iostream>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include "device_functions.cuh"
#include "kernel_functions.cuh"
#include <stdio.h>
#include "VTK.cu"
#include "utilities.cu"

//Initial conditions
const float PARTICLE_RADIUS = 1/10.f;
const float mass = M_PI * pow(PARTICLE_RADIUS,3)/3*4;
const float PARTICLE_DIAMETER = 2 * PARTICLE_RADIUS;
const float STARTING_POSITION[3] = { 0,0,0 };
const float FINAL_POSITION[3] = { 1,1,1 };
int NPD[3];
float VOLUME = 1;
const int SIMULATION_DIMENSION = 3;
const int x = 40; // Number of particles inside the smoothing length

int iteration = 0;
float simulation_time = 0;

//Value for PI -> M_PI

int main(void)
{
    // Get number per dimension (NPD) of particles
    for (int i = 0; i < 3; i++) {
        NPD[i] = ceil((FINAL_POSITION[i] - STARTING_POSITION[i]) / PARTICLE_DIAMETER);
        VOLUME = VOLUME * (FINAL_POSITION[i] - STARTING_POSITION[i]);
    }
    
    int N = NPD[0] * NPD[1] * NPD[2];
    int SIM_SIZE = N * SIMULATION_DIMENSION;
    const float h = pow(3 * VOLUME * x/(4*M_PI*N),1/3.f);

    //const float boundary_radius = h/4;
    //const float boundary_diameter = h/2;

    //printf("%g\n",h);

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);

    vec3d* POSITIONS;

    // Allocate Unified Memory accessible from CPU or GPU
    hipMallocManaged(&POSITIONS, SIM_SIZE * sizeof(float));

    // Define grid and block allocations for CUDA kernel function
    dim3 block(1, 1, 1);
    dim3 grid(NPD[0], NPD[1], NPD[2]);
    
    //generate locations for each particle
    getPositions<<<grid,block>>>(POSITIONS, PARTICLE_DIAMETER, SIMULATION_DIMENSION, SIM_SIZE);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    hipDeviceSynchronize();

    float* density = new float[N];
    for (int i = 0; i < N; i++){
        density[i] = 1000;
    }

    vec3d* velocity = new vec3d[N];
    for (int i = 0; i < N; i++){
        velocity[i].x = i;
        velocity[i].y = i;
        velocity[i].z = i;
    }

    float** pointData[] = {&density,&density,&density};
    int size_pointData = sizeof(pointData)/sizeof(double);
    vec3d** vectorData[] = {&velocity,&velocity};
    int size_vectorData = sizeof(vectorData)/sizeof(double);
    // std::cout << sizeof(vectorData) << std::endl;
    // std::cout << typeid(vectorData).name() << std::endl;
    std::string pointDataNames[] = {"density","density2","density3"};
    std::string vectorDataNames[] = {"velocity","velocity2"};

    char vtu_path[] = "results";

    CreateDir(vtu_path);

    VTU_Writer(vtu_path,iteration,POSITIONS,N,pointData,vectorData,pointDataNames,vectorDataNames,size_pointData,size_vectorData);

    // Free memory
    hipFree(POSITIONS);

    return 0;
}