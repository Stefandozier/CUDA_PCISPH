#include "common.cuh"
#include "global_variables.cuh"

int initialize();

int mainLoop();

void multiprocessor_writer();

int main(void)
{
	int init = initialize();

	if (init != 0) {
		printf("\n\nINITIALIZATION ERROR\n\n");
		return 1;
	}

	while (simulation_time < final_time)
	{
		int main_loop = mainLoop();

		if (main_loop != 0) {
			printf("\n\nMAIN LOOP ERROR\n\n");
			return 1;
		}

		multiprocessor_writer();
		simulation_time = final_time;
	}
	

	hipDeviceReset();

	return 0;
}