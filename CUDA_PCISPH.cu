#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include "particle_positions.cuh"
#include "utilities.cuh"
#include "VTK.cuh"
#include "hashing.cuh"
#include "particle_parameters.cuh"

//float inf = std::numeric_limits<float>::infinity();

// Initial conditions
const float PARTICLE_RADIUS = 0.01f;
const float MASS = (float)M_PI *- pow(PARTICLE_RADIUS, 3.f) / 3.f * 4.f;
const float PARTICLE_DIAMETER = 2 * PARTICLE_RADIUS;
const float F_INITIAL_POSITION[3] = { -0.5,-0.5,-0.5 }; //Fluid particles initial position
const float F_FINAL_POSITION[3] = { 0.5,0.5,0.5 }; //Fluid particles final position
const float B_INITIAL_POSITION[3] = { -0.5,-0.5,-0.5 }; //Boundary particles final position
const float B_FINAL_POSITION[3] = { 0.5,0.5,0.5 }; //Boundary particles final position
float VOLUME = 1;
const int SIMULATION_DIMENSION = 3;
const int x = 40; // Number of particles inside the smoothing length

int iteration = 1;
float simulation_time = 0;

// Value for PI -> M_PI

int hashFunction(vec3d point, float h,int hashtable_size) {

	int r_x, r_y, r_z;

	r_x = static_cast<int>((point.x / h)) * 73856093;
	r_y = static_cast<int>((point.y / h)) * 19349669;
	r_z = static_cast<int>((point.z / h)) * 83492791;
	//printf("[%g %g %g] -> %d\n", point.x, point.y, point.z, (r_x ^ r_y ^ r_z) & this->hashtable_size);
	//printf("%d %d\n", (r_x ^ r_y ^ r_z), this->hashtable_size);
	return ((r_x ^ r_y ^ r_z) & hashtable_size) - 1;
	}

int main(void)
{
	int block_size = 1024;
	// get main path of simulation
	char main_path[1024];
	getMainPath(main_path);

	// write path for vtu files
	char vtu_path[1024];
	strcpy(vtu_path, main_path);
	strcat(vtu_path, "/vtu");

	// write path for vtk group file
	char vtk_group_path[1024];
	strcpy(vtk_group_path, main_path);
	strcat(vtk_group_path, "/PCISPH.pvd");

	// create directory for vtu files
	CreateDir(vtu_path);

	// Get number per dimension (NPD) of FLUID particles for hexadecimal packing (assuming use of makeprism function)
	int NPD[3];
	for (int i = 0; i < 3; i++) {
		if (i == 1) {
			NPD[i] = floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / (sqrt(3.f) / 2.f * PARTICLE_DIAMETER));
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
		else {
			NPD[i] = floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / PARTICLE_DIAMETER);
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
	}

	//Passing NPD to device
	int* D_NPD;
	gpuErrchk(hipMalloc((void**)&D_NPD,SIMULATION_DIMENSION*sizeof(float)));
	gpuErrchk(hipMemcpy(D_NPD, NPD, SIMULATION_DIMENSION * sizeof(float), hipMemcpyHostToDevice));

	int N = NPD[0] * NPD[1] * NPD[2]; //number of fluid particles
	int SIM_SIZE = N * SIMULATION_DIMENSION;
	const float h = pow(3.f * VOLUME * x / (4.f * M_PI * N), 1.f / 3.f);

	//defining gravity vector
	gravity.x = 0;
	gravity.y = -9.81;
	gravity.z = 0;

	const float boundary_radius = h/4;
	const float boundary_diameter = h/2;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	vec3d f_initial;
	f_initial.x = F_INITIAL_POSITION[0] + PARTICLE_RADIUS;
	f_initial.y = F_INITIAL_POSITION[1] + PARTICLE_RADIUS;
	f_initial.z = F_INITIAL_POSITION[2] + PARTICLE_RADIUS;

	size_t bytes_fluid_particles = SIM_SIZE * sizeof(float);

	vec3d* FLUID_POSITIONS; //host pointer
	FLUID_POSITIONS = (vec3d*)malloc(bytes_fluid_particles);

	vec3d* D_FLUID_POSITIONS; //device pointer
	gpuErrchk(hipMalloc((void**)&D_FLUID_POSITIONS, bytes_fluid_particles));

	// grid -> number of blocks
	// block -> number of threads

	int grid_size = N/ block_size + 1;

	//generate locations for each particle
	makePrism << <grid_size, block_size >> > (D_FLUID_POSITIONS, PARTICLE_DIAMETER, f_initial, D_NPD, N);

	// Get number per dimension (NPD) of BOUNDARY particles without compact packing (assuming use of makebox function)
	for (int i = 0; i < 3; i++) {
		NPD[i] = ceil((B_FINAL_POSITION[i] - B_INITIAL_POSITION[i]) / PARTICLE_DIAMETER) + 2;
	}

	//copy new NPD to device memory
	gpuErrchk(hipMemcpy(D_NPD, NPD, SIMULATION_DIMENSION * sizeof(float), hipMemcpyHostToDevice));

	int B = NPD[0] * NPD[1] * NPD[2] - (NPD[0] - 2) * (NPD[1] - 2) * (NPD[2] - 2); //Number of boundary particles
	SIM_SIZE = NPD[0] * NPD[1] * NPD[2] * SIMULATION_DIMENSION;

	vec3d b_initial;
	b_initial.x = B_INITIAL_POSITION[0] - PARTICLE_RADIUS;
	b_initial.y = B_INITIAL_POSITION[1] - PARTICLE_RADIUS;
	b_initial.z = B_INITIAL_POSITION[2] - PARTICLE_RADIUS;
	vec3d b_final;
	b_final.x = b_initial.x + PARTICLE_DIAMETER * (NPD[0] - 1);
	b_final.y = b_initial.y + PARTICLE_DIAMETER * (NPD[1] - 1);
	b_final.z = b_initial.z + PARTICLE_DIAMETER * (NPD[2] - 1);

	size_t bytes_boundary_particles = SIM_SIZE * sizeof(float);
	vec3d* BOUNDARY_POSITIONS; //host pointer
	BOUNDARY_POSITIONS = (vec3d*)malloc(bytes_boundary_particles); //allocate memory in the host

	vec3d* D_BOUNDARY_POSITIONS; //device pointer
	gpuErrchk(hipMalloc((void**)&D_BOUNDARY_POSITIONS, bytes_boundary_particles)); // allocate memory in the device

	makeBox(D_BOUNDARY_POSITIONS, PARTICLE_DIAMETER, b_initial, b_final, block_size,D_NPD);

	int T = N + B; //Total number of particles

	std::cout << "Initializing with " << N << " fluid particles and " << B << " boundary particles.\n"
		<< "Total of " << T << " particles.\n"
		<< "Smoothing radius = " << h << " m.\n";

	float* density = new float[N];
	for (int i = 0; i < N; i++) {
		density[i] = 1000;
	}

	vec3d* velocity = new vec3d[N];
	for (int i = 0; i < N; i++) {
		velocity[i].x = i;
		velocity[i].y = i;
		velocity[i].z = i;
	}

	float** pointData[] = { &density };
	int size_pointData = sizeof(pointData) / sizeof(double);
	vec3d** vectorData[] = { &velocity };
	int size_vectorData = sizeof(vectorData) / sizeof(double);

	std::string pointDataNames[] = { "density" };
	std::string vectorDataNames[] = { "velocity" };

	char vtu_fullpath[1024];
	// hipDeviceSynchronize();

	gpuErrchk(hipMemcpy(FLUID_POSITIONS, D_FLUID_POSITIONS, bytes_fluid_particles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(BOUNDARY_POSITIONS, D_BOUNDARY_POSITIONS, bytes_boundary_particles, hipMemcpyDeviceToHost));

	// Free GPU memory for fluid particles
	hipFree(D_FLUID_POSITIONS);

	// HASHING ONLY FOR BOUNDARY PARTICLES
	const int hashtable_size = nextPrime(2*B) + 1;

	Hash hash(hashtable_size);
	const int particles_per_row = 200;
	size_t pitch = 0;
	int* hashtable = new int[hashtable_size * particles_per_row];
	for (int i = 0; i < hashtable_size; ++i) {
		for (int j = 0; j < particles_per_row; j++) {
			hashtable[i * particles_per_row + j] = -1;
		}
	}

	int* d_hashtable;

	size_t width = particles_per_row * sizeof(int);
	size_t height = hashtable_size;

	gpuErrchk(hipMallocPitch(&d_hashtable, &pitch, particles_per_row * sizeof(int), hashtable_size));
	gpuErrchk(hipMemcpy2D(d_hashtable, pitch, hashtable, particles_per_row * sizeof(int), width, height, hipMemcpyHostToDevice));

	grid_size = B / block_size + 1;
	hashParticlePositions << <grid_size, block_size >> > (d_hashtable, D_BOUNDARY_POSITIONS, h, hash, B, pitch, particles_per_row);

	float* d_boundary_mass;
	gpuErrchk(hipMalloc((void**)&d_boundary_mass, B * sizeof(float)));

	boundaryPsi << <grid_size, block_size >> > (d_boundary_mass, d_hashtable, rho_0, D_BOUNDARY_POSITIONS, h, particles_per_row, pitch, hash, B);

	float *boundary_mass = (float*)malloc(B*sizeof(float));
	gpuErrchk(hipMemcpy(boundary_mass, d_boundary_mass,(size_t)B * sizeof(float), hipMemcpyDeviceToHost));

	float** boundary_point_data[] = { &boundary_mass };
	size_pointData = sizeof(pointData) / sizeof(double);
	vec3d** boundary_vectorData[1] = { };
	size_vectorData = 0;

	std::string boundary_pointDataNames[] = { "psi" };
	std::string boundary_vectorDataNames[1] = {  };

	VTU_Writer(main_path, iteration, BOUNDARY_POSITIONS, B, boundary_point_data, boundary_vectorData, boundary_pointDataNames, boundary_vectorDataNames, size_pointData, size_vectorData, vtu_fullpath, 1);

	//gpuErrchk(hipMemcpy2D(hashtable, particles_per_row * sizeof(int), d_hashtable, pitch, width, height, hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	//END OF HASHING FOR BOUNDARIES

	hipFree(d_hashtable);

	hipFree(D_BOUNDARY_POSITIONS);

	strcpy(vtu_fullpath, VTU_Writer(vtu_path, iteration, FLUID_POSITIONS, N, pointData, vectorData, pointDataNames, vectorDataNames, size_pointData, size_vectorData, vtu_fullpath));

	VTK_Group(vtk_group_path, vtu_fullpath, simulation_time);

	return 0;
}