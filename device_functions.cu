//All functions only callable from the device.

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


__device__ int getGlobalIdx_3D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
};

__device__ float Poly6_Kernel(float r,float h,float pi)
{
    return 315/(64*pi*powf(pi,9))*powf(powf(r,2)-powf(r,2),3);
}