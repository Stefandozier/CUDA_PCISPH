#include "hip/hip_runtime.h"
#include "particle_positions.cuh"
#include "utilities.cuh"
#include "VTK.cuh"
#include "hashing.cuh"
#include "particle_parameters.cuh"
#include "global_variables.cuh"
#include <math.h>
#include <future> 
#include <chrono>

//declaration of all global variables that are going to be used in this file
char main_path[1024];
char vtk_group_path[1024];
char vtu_fullpath[1024];
char vtu_path[1024];
float** pointData[2];
vec3d** vectorData[5];
std::string pointDataNames[2];
std::string vectorDataNames[5];
int size_pointData;
int size_vectorData;

vec3d* POSITION;
vec3d* d_POSITION;
vec3d* VELOCITY;
vec3d* d_VELOCITY;
vec3d* ST_FORCE;
vec3d* d_ST_FORCE;
vec3d* VISCOSITY_FORCE;
vec3d* d_VISCOSITY_FORCE;
vec3d* PRESSURE_FORCE;
vec3d* d_PRESSURE_FORCE;
vec3d* NORMAL;
vec3d* d_NORMAL;
float* DENSITY;
float* d_DENSITY;
float* PRESSURE;
float* d_PRESSURE;
float* MASS;
float* d_MASS;
int* TYPE;
int* d_TYPE;
int* hashtable;
int* d_hashtable;
vec3d gravity;

//initial conditions
const float PARTICLE_RADIUS = 0.011f;
const float MASS_calc = (float)M_PI * -pow(PARTICLE_RADIUS, 3.f) / 3.f * 4.f;
const float PARTICLE_DIAMETER = 2 * PARTICLE_RADIUS;
const float F_INITIAL_POSITION[3] = { 0.f,0.f,0.f }; //Fluid particles initial position
const float F_FINAL_POSITION[3] = { 0.5f,1.f,0.5f }; //Fluid particles final position
const float B_INITIAL_POSITION[3] = { 0.f,0.f,0.f }; //Boundary particles final position
const float B_FINAL_POSITION[3] = { 1.f,1.f,1.f }; //Boundary particles final position

//physical constants
const float rho_0 = 1000.f;
const float visc_const = 0.0010518f;
const float st_const = 0.0728f;

//controlling iteration number and simulation time
int iteration = 1;
float simulation_time = 0.f; //in seconds
float final_time = 10.f; //in seconds

//number of particles
int N; //fluid particles
int B; //bondary particles
int T; //total particles

//variables for hashtable
size_t pitch;
const int particles_per_row = 200;
int hashtable_size;
//const int n_p_neighbors = 8000; //in case of memory failure, raise this number

//simulation parameters
float invh;
float h;

//CUDA variables
int block_size = 1024;
int grid_size;

int initialize() {

	// get main path of simulation

	getMainPath(main_path);

	// write path for vtu files
	strcpy(vtu_path, main_path);
	strcat(vtu_path, "/vtu");

	// write path for vtk group file

	strcpy(vtk_group_path, main_path);
	strcat(vtk_group_path, "/PCISPH.pvd");

	// create directory for vtu files
	CreateDir(vtu_path);

	float VOLUME = 1;
	const int SIMULATION_DIMENSION = 3;

	// Get number per dimension (NPD) of FLUID particles for hexadecimal packing (assuming use of makeprism function)
	int NPD[3];
	for (int i = 0; i < 3; i++) {
		if (i == 1) {
			NPD[i] = static_cast<int>(floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / (sqrt(3.f) / 2.f * PARTICLE_DIAMETER)));
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
		else {
			NPD[i] = static_cast<int>(floor((F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]) / PARTICLE_DIAMETER));
			VOLUME = VOLUME * (F_FINAL_POSITION[i] - F_INITIAL_POSITION[i]);
		}
	}

	//Passing NPD to device
	int* D_NPD;
	gpuErrchk(hipMalloc((void**)&D_NPD, SIMULATION_DIMENSION * sizeof(float)));
	gpuErrchk(hipMemcpy(D_NPD, NPD, SIMULATION_DIMENSION * sizeof(float), hipMemcpyHostToDevice));

	N = NPD[0] * NPD[1] * NPD[2]; //number of fluid particles
	int SIM_SIZE = N * SIMULATION_DIMENSION;
	const int x = 40; // Number of particles inside the smoothing length
	h = powf(3.f * VOLUME * x / (4.f * (float)M_PI * N), 1.f / 3.f);
	//h = 0.02;
	invh = 1 / h;

	//const float boundary_radius = h/4;
	//const float boundary_diameter = h/2;

	//hipError_t cudaStatus;
	//cudaStatus = hipSetDevice(0);

	vec3d f_initial;
	f_initial.x = F_INITIAL_POSITION[0] + PARTICLE_RADIUS;
	f_initial.y = F_INITIAL_POSITION[1] + PARTICLE_RADIUS;
	f_initial.z = F_INITIAL_POSITION[2] + PARTICLE_RADIUS;

	size_t bytes_fluid_particles = SIM_SIZE * sizeof(float);

	vec3d* FLUID_POSITIONS; //host pointer
	FLUID_POSITIONS = (vec3d*)malloc(bytes_fluid_particles);

	vec3d* D_FLUID_POSITIONS; //device pointer
	gpuErrchk(hipMalloc((void**)&D_FLUID_POSITIONS, bytes_fluid_particles));

	// grid -> number of blocks
	// block -> number of threads

	grid_size = N / block_size + 1;

	//generate locations for each particle
	makePrism << <grid_size, block_size >> > (D_FLUID_POSITIONS, PARTICLE_DIAMETER, f_initial, D_NPD, N);

	float BOUNDARY_DIAMETER = h/2;
	float BOUNDARY_RADIUS = h/4;

	// Get number per dimension (NPD) of BOUNDARY particles without compact packing (assuming use of makebox function)
	for (int i = 0; i < 3; i++) {
		NPD[i] = static_cast<int>(ceil((B_FINAL_POSITION[i] - B_INITIAL_POSITION[i]) / BOUNDARY_DIAMETER)) + 2;

	}

	//copy new NPD to device memory
	gpuErrchk(hipMemcpy(D_NPD, NPD, SIMULATION_DIMENSION * sizeof(float), hipMemcpyHostToDevice));

	B = NPD[0] * NPD[1] * NPD[2] - (NPD[0] - 2) * (NPD[1] - 2) * (NPD[2] - 2); //Number of boundary particles
	SIM_SIZE = NPD[0] * NPD[1] * NPD[2] * SIMULATION_DIMENSION;

	vec3d b_initial;
	b_initial.x = B_INITIAL_POSITION[0] - BOUNDARY_RADIUS;
	b_initial.y = B_INITIAL_POSITION[1] - BOUNDARY_RADIUS;
	b_initial.z = B_INITIAL_POSITION[2] - BOUNDARY_RADIUS;
	vec3d b_final;
	b_final.x = b_initial.x + BOUNDARY_DIAMETER * (NPD[0] - 1);
	b_final.y = b_initial.y + BOUNDARY_DIAMETER * (NPD[1] - 1);
	b_final.z = b_initial.z + BOUNDARY_DIAMETER * (NPD[2] - 1);

	//printf("[%g %g %g] [%g %g %g]\n", b_final.x, b_final.y, b_final.z, B_FINAL_POSITION[0] + BOUNDARY_RADIUS, B_FINAL_POSITION[1] + BOUNDARY_RADIUS, B_FINAL_POSITION[2] + BOUNDARY_RADIUS);

	size_t bytes_boundary_particles = SIM_SIZE * sizeof(float);
	vec3d* BOUNDARY_POSITIONS; //host pointer
	BOUNDARY_POSITIONS = (vec3d*)malloc(bytes_boundary_particles); //allocate memory in the host

	vec3d* D_BOUNDARY_POSITIONS; //device pointer
	gpuErrchk(hipMalloc((void**)&D_BOUNDARY_POSITIONS, bytes_boundary_particles)); // allocate memory in the device

	makeBox(D_BOUNDARY_POSITIONS, BOUNDARY_DIAMETER, b_initial, b_final, block_size, D_NPD);

	T = N + B; //Total number of particles

	gpuErrchk(hipMemcpy(FLUID_POSITIONS, D_FLUID_POSITIONS, bytes_fluid_particles, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(BOUNDARY_POSITIONS, D_BOUNDARY_POSITIONS, bytes_boundary_particles, hipMemcpyDeviceToHost));

	// Free GPU memory for fluid particles
	hipFree(D_FLUID_POSITIONS);

	// HASHING ONLY FOR BOUNDARY PARTICLES
	hashtable_size = nextPrime(2*B) + 1;

	Hash b_hash(hashtable_size);
	const int particles_per_row = 200;
	pitch = 0;
	hashtable = new int[hashtable_size * particles_per_row];
	for (int i = 0; i < hashtable_size; ++i) {
		for (int j = 0; j < particles_per_row; j++) {
			hashtable[i * particles_per_row + j] = -1;
		}
	}

	gpuErrchk(hipMallocPitch(&d_hashtable, &pitch, particles_per_row * sizeof(int), hashtable_size));
	gpuErrchk(hipMemcpy2D(d_hashtable, pitch, hashtable, particles_per_row * sizeof(int), particles_per_row * sizeof(int), hashtable_size, hipMemcpyHostToDevice));

	grid_size = B / block_size + 1;
	hashParticlePositions << <grid_size, block_size >> > (d_hashtable, D_BOUNDARY_POSITIONS, invh, b_hash, B, pitch, particles_per_row);

	// Calculate mass (or psi) for each boundary particle

	float* d_boundary_mass;
	gpuErrchk(hipMalloc((void**)&d_boundary_mass, B * sizeof(float)));

	boundaryPsi << <grid_size, block_size >> > (d_boundary_mass, d_hashtable, rho_0, D_BOUNDARY_POSITIONS, h, invh, particles_per_row, pitch, b_hash, B);

	float* boundary_mass = (float*)malloc(B * sizeof(float));
	gpuErrchk(hipMemcpy(boundary_mass, d_boundary_mass, (size_t)B * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_boundary_mass));

	//Calculate normal for boundary particles

	vec3d* d_boundary_normal;
	gpuErrchk(hipMalloc((void**)&d_boundary_normal, B * 3 * sizeof(float)));

	boundaryNormal << <grid_size, block_size >> > (d_boundary_normal, D_BOUNDARY_POSITIONS, b_initial, b_final, B);

	vec3d* boundary_normal = (vec3d*)malloc(B * 3 * sizeof(float));
	gpuErrchk(hipMemcpy(boundary_normal, d_boundary_normal, (size_t)B * 3 * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_boundary_normal));

	//Write boundary vtu file

	float** boundary_point_data[] = { &boundary_mass };
	size_pointData = sizeof(boundary_point_data) / sizeof(double);
	vec3d** boundary_vectorData[] = { &boundary_normal };
	size_vectorData = sizeof(boundary_vectorData) / sizeof(double);

	std::string boundary_pointDataNames[] = { "psi" };
	std::string boundary_vectorDataNames[] = { "normal" };

	VTU_Writer(main_path, iteration, BOUNDARY_POSITIONS, B, boundary_point_data, boundary_vectorData, boundary_pointDataNames, boundary_vectorDataNames, size_pointData, size_vectorData, vtu_fullpath, 1);

	//gpuErrchk(hipMemcpy2D(hashtable, particles_per_row * sizeof(int), d_hashtable, pitch, width, height, hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	//END OF HASHING FOR BOUNDARIES

	hipFree(d_hashtable);

	hipFree(D_BOUNDARY_POSITIONS);

	//Initializing main particle variables

	//Defining and allocating main position variable
	
	POSITION = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < N; i++) {
		POSITION[i].x = FLUID_POSITIONS[i].x;
		POSITION[i].y = FLUID_POSITIONS[i].y;
		POSITION[i].z = FLUID_POSITIONS[i].z;
	}

	for (int i = N; i < T; i++) {
		POSITION[i].x = BOUNDARY_POSITIONS[i - N].x;
		POSITION[i].y = BOUNDARY_POSITIONS[i - N].y;
		POSITION[i].z = BOUNDARY_POSITIONS[i - N].z;
	}

	free(BOUNDARY_POSITIONS);
	free(FLUID_POSITIONS);

	
	gpuErrchk(hipMalloc((void**)&d_POSITION, 3*T*sizeof(float)));
	gpuErrchk(hipMemcpy(d_POSITION, POSITION, 3*T*sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main velocity variable
	
	VELOCITY = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < T; i++) {
		VELOCITY[i].x = 0.f;
		VELOCITY[i].y = 0.f;
		VELOCITY[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&d_VELOCITY, 3*T*sizeof(float)));
	gpuErrchk(hipMemcpy(d_VELOCITY, VELOCITY, 3*T*sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main st force variable
	
	ST_FORCE = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < T; i++) {
		ST_FORCE[i].x = 0.f;
		ST_FORCE[i].y = 0.f;
		ST_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&d_ST_FORCE, 3*T*sizeof(float)));

	//Defining and allocating main viscosity force variable
	VISCOSITY_FORCE = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < T; i++) {
		VISCOSITY_FORCE[i].x = 0.f;
		VISCOSITY_FORCE[i].y = 0.f;
		VISCOSITY_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&d_VISCOSITY_FORCE, 3*T*sizeof(float)));

	//Defining and allocating main pressure force variable
	PRESSURE_FORCE = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < T; i++) {
		PRESSURE_FORCE[i].x = 0.f;
		PRESSURE_FORCE[i].y = 0.f;
		PRESSURE_FORCE[i].z = 0.f;
	}

	gpuErrchk(hipMalloc((void**)&d_PRESSURE_FORCE, 3*T*sizeof(float)));

	//Defining and allocating main normal variable
	NORMAL = (vec3d*)malloc(3*T*sizeof(float));
	for (int i = 0; i < N; i++) {
		NORMAL[i].x = 0.f;
		NORMAL[i].y = 0.f;
		NORMAL[i].z = 0.f;
	}	

	for (int i = N; i < T; i++) {
		NORMAL[i].x = boundary_normal[i - N].x;
		NORMAL[i].y = boundary_normal[i - N].y;
		NORMAL[i].z = boundary_normal[i - N].z;
	}

	free(boundary_normal);

	gpuErrchk(hipMalloc((void**)&d_NORMAL, 3*T*sizeof(float)));
	gpuErrchk(hipMemcpy(d_NORMAL, NORMAL, 3*T*sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main density array
	DENSITY = (float*)malloc(T * sizeof(float));
	for (int i = 0; i < T; i++) {
		DENSITY[i] = rho_0;
	}

	gpuErrchk(hipMalloc((void**)&d_DENSITY, T * sizeof(float)));
	gpuErrchk(hipMemcpy(d_DENSITY, DENSITY, T * sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main pressure array
	PRESSURE = (float*)malloc(T * sizeof(float));
	for (int i = 0; i < T; i++) {
		PRESSURE[i] = 0;
	}

	gpuErrchk(hipMalloc((void**)&d_PRESSURE, T * sizeof(float)));

	//Defining and allocating main mass array
	
	MASS = (float*)malloc(T * sizeof(float));
	for (int i = 0; i < N; i++) {
		MASS[i] = MASS_calc;
	}

	for (int i = N; i < T; i++) {
		MASS[i] = boundary_mass[i - N];
	}

	free(boundary_mass);
	
	gpuErrchk(hipMalloc((void**)&d_MASS, T * sizeof(float)));
	gpuErrchk(hipMemcpy(d_MASS, MASS, T * sizeof(float), hipMemcpyHostToDevice));

	//Defining and allocating main type array (0 if fluid, 1 if boundary)
	TYPE = (int*)malloc(T * sizeof(int));
	for (int i = 0; i < N; i++) {
		TYPE[i] = 0;
	}

	for (int i = N; i < T; i++) {
		TYPE[i] = 1;
	}

	gpuErrchk(hipMalloc((void**)&d_TYPE, T * sizeof(int)));
	gpuErrchk(hipMemcpy(d_TYPE, TYPE, T * sizeof(int), hipMemcpyHostToDevice));

	//defining gravity vector
	gravity.x = 0.f;
	gravity.y = 9.81f;
	gravity.z = 0.f;

	//Defining variables to write VTU files
	pointData[0] = &DENSITY;
	pointData[1] = &PRESSURE;
	size_pointData = sizeof(pointData) / sizeof(double);

	vectorData[0] = &VELOCITY;
	vectorData[1] = &PRESSURE_FORCE;
	vectorData[2] = &VISCOSITY_FORCE;
	vectorData[3] = &ST_FORCE;
	vectorData[4] = &NORMAL;
	size_vectorData = sizeof(vectorData) / sizeof(double);

	pointDataNames[0] = "density";
	pointDataNames[1] = "pressure";
	vectorDataNames[0] = "velocity";
	vectorDataNames[1] = "pressure force";
	vectorDataNames[2] = "viscosity force";
	vectorDataNames[3] = "st force";
	vectorDataNames[4] = "normal";

	auto started = std::chrono::high_resolution_clock::now();

	VTU_Writer(vtu_path, iteration, POSITION, N, pointData, vectorData, pointDataNames, vectorDataNames, size_pointData, size_vectorData, vtu_fullpath);

	auto done = std::chrono::high_resolution_clock::now();

	std::cout << "First VTU_Writer() -> " << std::chrono::duration_cast<std::chrono::milliseconds>(done - started).count() << " ms\n";

	VTK_Group(vtk_group_path, vtu_fullpath, simulation_time);

	// Initialize main hashtable

	hashtable_size = nextPrime(2 * T) + 1;

	hashtable = new int[hashtable_size * particles_per_row];
	for (int i = 0; i < hashtable_size; ++i) {
		for (int j = 0; j < particles_per_row; j++) {
			hashtable[i * particles_per_row + j] = -1;
		}
	}

	gpuErrchk(hipMallocPitch(&d_hashtable, &pitch, particles_per_row * sizeof(int), hashtable_size));
	gpuErrchk(hipMemcpy2D(d_hashtable, pitch, hashtable, particles_per_row * sizeof(int), particles_per_row * sizeof(int), hashtable_size, hipMemcpyHostToDevice));

	std::cout << "Initializing with " << N << " fluid particles and " << B << " boundary particles.\n"
		<< "Total of " << T << " particles.\n"
		<< "Smoothing radius = " << h << " m.\n"
		<< "hashtable size = " << hashtable_size << "\n";

	return 0;
}

int mainLoop() {

	Hash hash(hashtable_size);

	grid_size = T / block_size + 1;
	hashParticlePositions << <grid_size, block_size >> > (d_hashtable, d_POSITION, invh, hash, T, pitch, particles_per_row);
	gpuErrchk(hipDeviceSynchronize());
	printf("hashing done\n");

	grid_size = N / block_size + 1;
	fluidNormal << <grid_size, block_size >> > (d_NORMAL, d_POSITION, d_MASS, d_DENSITY, h,invh, hash,d_hashtable, particles_per_row,pitch, N);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipPeekAtLastError());
	nonPressureForces << <grid_size, block_size >> > (d_POSITION, d_VISCOSITY_FORCE, d_ST_FORCE, d_MASS, d_DENSITY, d_VELOCITY, d_NORMAL, gravity, h, invh, rho_0, visc_const, st_const, particles_per_row, pitch,d_hashtable, hash, N);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipMemcpy(NORMAL, d_NORMAL, N * 3 * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(VISCOSITY_FORCE, d_VISCOSITY_FORCE, N * 3 * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(ST_FORCE, d_ST_FORCE, N * 3 * sizeof(float), hipMemcpyDeviceToHost));

	iteration++;

	return 0;
}

void multiprocessor_writer() {

	char buf[1024];
	itoa(iteration, buf, 10);
	strcpy(vtu_fullpath, vtu_path);
	strcat(vtu_fullpath, "/iter");
	strcat(vtu_fullpath, buf);
	strcat(vtu_fullpath, ".vtu");

	std::future<void> write_vtu;

	auto started = std::chrono::high_resolution_clock::now();

	try {
		write_vtu.wait();
	}
	catch (std::exception& e) {
		//DO NOTHING
	}

	write_vtu = std::async(std::launch::async, VTU_Writer, vtu_path, iteration, POSITION, N, pointData, vectorData, pointDataNames, vectorDataNames, size_pointData, size_vectorData, vtu_fullpath,2);
	auto done = std::chrono::high_resolution_clock::now();

	std::cout << "Second VTU_Writer() -> " << std::chrono::duration_cast<std::chrono::milliseconds>(done - started).count() << " ms\n";
	strcpy(buf, vtu_fullpath);

	VTK_Group(vtk_group_path, buf, simulation_time);
	//write_vtu.get();
	return;
}